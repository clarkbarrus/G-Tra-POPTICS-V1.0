#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>

#include <iostream>
#include <vector>
#include <thrust/device_vector.h>

#include "trajectory_data.cuh"
#include "g_tra_poptics.cuh"
#include "rtree.h"
#include "strtree.cuh"

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

/**
 *
 * Entry point for executing G-Tra-POPTICS
 */
int main(int argc, char **argv)
{

	std::string file_name = "testtrajectorydata.csv";

	// Load trajectory data from file
	file_trajectory_data trajectory_data = load_trajectory_data_from_file(file_name);

	// Preprocessing: build STR-tree index
	thrust::host_vector<strtree_line> lines = points_to_line_vector(trajectory_data.points, trajectory_data.trajectories,
			trajectory_data.num_points, trajectory_data.num_trajectories);
	strtree strtree = cuda_create_strtree(lines);

	thrust::host_vector<strtree_offset_node> nodes = strtree.nodes;

	for(int i = 0; i < nodes.size(); i++)
	{
		strtree_offset_node node = nodes[i];
		std::cout << "Node " << i << ": num children=" << node.num << ", depth=" << node.depth << ", child_offset=" <<node.first_child_offset
			<< ", bbox.x1=" << node.boundingbox.x1 << ", bbox.x2=" << node.boundingbox.x2
			<< ", bbox.y1=" << node.boundingbox.y1 << ", bbox.y2=" << node.boundingbox.y2
			<< ", bbox.t1=" << node.boundingbox.t1 << ", bbox.t2=" << node.boundingbox.t2
			<< std::endl;
	}

//	/* Initialize variables for G-Tra-POPTICS execution */
//	// Number of CPU threads executing
//	int cpu_threads = 8;
//	// Maximum epsilon at which clusters are detected
//	double epsilon = 0.2;
//	// Specific epsilon for which to find clusters after minimum spanning trees are built
//	double epsilon_prime = 0.1;
//	// Minimum number of trajectories near a point for it to be considered a core point.
//	double min_num_trajectories = 2;
//
//	// Execute G-Tra-POPTICS on data file
//	g_tra_poptics(strtree, cpu_threads, epsilon, epsilon_prime, min_num_trajectories);

	return 0;
}

